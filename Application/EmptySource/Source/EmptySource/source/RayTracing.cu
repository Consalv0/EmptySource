#include "hip/hip_runtime.h"

#include "..\Source\EmptySource\include\Core.h"
#include "..\Source\EmptySource\include\Graphics.h"
#include "..\Source\EmptySource\include\CoreTypes.h"

#include "..\Source\EmptySource\include\Texture2D.h"
#include "..\Source\EmptySource\include\Utility\CUDAUtility.h"
#include "..\Source\EmptySource\include\Utility\Timer.h"
#include "..\Source\EmptySource\include\Math\Math.h"
#include "..\Source\EmptySource\include\Mesh.h"

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <surface_functions.h>

surface<void, cudaSurfaceType2D> SurfaceWrite;

__global__ void WirteTextureKernel(int2 TextureDimension) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= TextureDimension.x || y >= TextureDimension.y) return;

	uchar4 element = make_uchar4(0, 0, 0, 0);
	surf2Dread(&element, SurfaceWrite, x * sizeof(uchar4), y);
	element = make_uchar4(
		(float(x) / float(TextureDimension.x)) * 255.99F,
		(float(y) / float(TextureDimension.y)) * 255.99F,
		0.2F * 255.99F,
		0
	);
	surf2Dwrite(element, SurfaceWrite, x * sizeof(uchar4), y);

}

extern "C"
void LaunchWriteTextureKernel(hipArray *cudaTextureArray, int2 TextureDim) {
	dim3 dimBlock(8, 8);
	dim3 dimGrid(TextureDim.x / dimBlock.x + 1, TextureDim.y / dimBlock.y + 1);

	hipError_t Error;

	// --- Bind voxel array to a writable CUDA surface
	Error = cudaBindSurfaceToArray(SurfaceWrite, cudaTextureArray);
	if (Error != hipSuccess) {
		Debug::Log(
			Debug::LogError, L"%s",
			CharToWChar(hipGetErrorString(Error))
		);
		return;
	}

	WirteTextureKernel <<< dimGrid, dimBlock >>> (TextureDim);

	Error = hipGetLastError();
	if (Error != hipSuccess) {
		Debug::Log(
			Debug::LogError, L"%s",
			CharToWChar(hipGetErrorString(Error))
		);
	}
}


int RayTracingTexture2D(Texture2D* texture) {
	hipGraphicsResource *cudaTextureResource;
	hipArray            *cudaTextureArray;

	CUDA::Check( hipProfilerStart() );

	// --- Register Image (texture) to CUDA Resource
	CUDA::Check( hipGraphicsGLRegisterImage(&cudaTextureResource,
		texture->GetTextureObject(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore) 
	);

	Debug::Timer Timer;
	Timer.Start();
	// --- Map CUDA resource
	CUDA::Check( hipGraphicsMapResources(1, &cudaTextureResource, 0) );
	{
		// --- Get mapped array
		CUDA::Check( hipGraphicsSubResourceGetMappedArray(&cudaTextureArray, cudaTextureResource, 0, 0) );
		IntVector2 TextureDim = texture->GetDimension();
		LaunchWriteTextureKernel(cudaTextureArray, { TextureDim.x, TextureDim.y });
	}
	CUDA::Check( hipGraphicsUnmapResources(1, &cudaTextureResource, 0) );

	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );

	Timer.Stop();
	IntVector2 TextureDim = texture->GetDimension();
	Debug::Log(
		Debug::LogDebug, L"CUDA Texture Write with total volume (%s): %dms",
		Text::FormatUnit(TextureDim.x * TextureDim.y, 0).c_str(),
		Timer.GetEnlapsed()
	);

	CUDA::Check( hipGraphicsUnregisterResource(cudaTextureResource) );

	return 0;
}