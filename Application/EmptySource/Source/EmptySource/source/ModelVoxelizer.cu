#include "hip/hip_runtime.h"

#include "..\Source\EmptySource\include\Core.h"
#include "..\Source\EmptySource\include\Graphics.h"
#include "..\Source\EmptySource\include\CoreTypes.h"

#include "..\Source\EmptySource\include\Utility\CUDAUtility.h"
#include "..\Source\EmptySource\include\Utility\Timer.h"
#include "..\Source\EmptySource\include\Math\Math.h"
#include "..\Source\EmptySource\include\Texture3D.h"
#include "..\Source\EmptySource\include\Mesh.h"

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <surface_functions.h>

surface<void, cudaSurfaceType3D> SurfaceWrite; 

__global__ void WirteKernel(int N, MeshVertex *Vertices, dim3 TextureDimension) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < N) {
		uint3 Position = make_uint3(Vertices[index].Position.x, Vertices[index].Position.y, Vertices[index].Position.z);
		if (Position.x >= TextureDimension.x || Position.y >= TextureDimension.y || Position.z >= TextureDimension.z) {
			return;
		}

		float4 element = make_float4(0.0F, 0.0F, 0.0F, 0.0F);
		surf3Dread(&element, SurfaceWrite, Position.x * sizeof(float4), Position.y, Position.z);
		element = make_float4(
			Vertices[index].Normal.x * 0.5F + element.x * 0.5F,
			Vertices[index].Normal.y * 0.5F + element.y * 0.5F,
			Vertices[index].Normal.z * 0.5F + element.z * 0.5F,
			0
		);
		surf3Dwrite(element, SurfaceWrite, Position.x * sizeof(float4), Position.y, Position.z);
	}
}

extern "C"
void LaunchWriteKernel(int N, MeshVertex * dVertices, hipArray *cudaTextureArray, dim3 TextureDim) {
	dim3 dimBlock(8, 8, 8);
	dim3 dimGrid(TextureDim.x / dimBlock.x, TextureDim.y / dimBlock.y, TextureDim.z / dimBlock.z);

	hipError_t Error;

	// --- Bind voxel array to a writable CUDA surface
	Error = cudaBindSurfaceToArray(SurfaceWrite, cudaTextureArray);
	if (Error != hipSuccess) {
		Debug::Log(
			Debug::LogError, L"%s",
			CharToWChar(hipGetErrorString(Error))
		);
		return;
	}

	WirteKernel<<< dimGrid, dimBlock >>> (N, dVertices, TextureDim);

	Error = hipGetLastError();
	if (Error != hipSuccess) {
		Debug::Log(
			Debug::LogError, L"%s",
			CharToWChar(hipGetErrorString(Error))
		);
	}
}


int VoxelizeToTexture3D(Texture3D* texture, int N, MeshVertex * Vertices) {
	hipGraphicsResource *cudaTextureResource;
	hipArray            *cudaTextureArray;
	MeshVertex           *dVertices;

	size_t Size = N * sizeof(MeshVertex);

	CUDA::Check( hipProfilerStart() );

	Debug::Timer Timer;
	Timer.Start();

	// --- Allocate Memory in Device
	CUDA::Check( hipMalloc(&dVertices, Size) );
	CUDA::Check( hipMemcpy(dVertices, Vertices, Size, hipMemcpyHostToDevice) );

	// --- Register Image (texture) to CUDA Resource
	CUDA::Check( hipGraphicsGLRegisterImage(&cudaTextureResource,
		texture->GetTextureObject(), GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore) 
	);

	Timer.Stop();
	Debug::Log(
		Debug::LogDebug, L"CUDA Host allocation of %s durantion: %dms",
		Text::FormatData((double)N * 2 * sizeof(MeshVertex) + Size, 2).c_str(),
		Timer.GetEnlapsed()
	);

	Timer.Start();
	// --- Map CUDA resource
	CUDA::Check( hipGraphicsMapResources(1, &cudaTextureResource, 0) );
	{
		// --- Get mapped array
		CUDA::Check( hipGraphicsSubResourceGetMappedArray(&cudaTextureArray, cudaTextureResource, 0, 0) );
		IntVector3 TextureDim = texture->GetDimension();
		LaunchWriteKernel(N, dVertices, cudaTextureArray, dim3(TextureDim.x, TextureDim.y, TextureDim.z));
	}
	CUDA::Check( hipGraphicsUnmapResources(1, &cudaTextureResource, 0) );

	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );

	Timer.Stop();
	IntVector3 TextureDim = texture->GetDimension();
	Debug::Log(
		Debug::LogDebug, L"CUDA Texture Write with total volume (%s): %dms",
		Text::FormatUnit(TextureDim.x * TextureDim.y * TextureDim.z, 0).c_str(),
		Timer.GetEnlapsed()
	);

	CUDA::Check( hipGraphicsUnregisterResource(cudaTextureResource) );

	return 0;
}