#include "hip/hip_runtime.h"

#include "..\Source\EmptySource\include\Core.h"
#include "..\Source\EmptySource\include\Graphics.h"
#include "..\Source\EmptySource\include\CoreTypes.h"

#include "..\Source\EmptySource\include\Texture2D.h"
#include "..\Source\EmptySource\include\Utility\CUDAUtility.h"
#include "..\Source\EmptySource\include\Utility\Timer.h"
#include "..\Source\EmptySource\include\Math\Math.h"
#include "..\Source\EmptySource\include\Mesh.h"

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <surface_functions.h>

surface<void, cudaSurfaceType2D> SurfaceWrite;

__device__ RayHit HitSphere(const Vector3& Center, const float& Radius, const float& MinDistance, const float& MaxDistance, const Ray& ray) {
	RayHit Hit;
	Vector3 OC = ray.Origin() - Center;
	float a = ray.Direction().Dot(ray.Direction());
	float b = OC.Dot(ray.Direction());
	float c = OC.Dot(OC) - Radius * Radius;
	float Discriminant = sqrtf((b * b) - (a * c));
	
	if (Discriminant >= 0.F) {
		float Stamp = (-b - Discriminant) / a;
		Vector3 Normal = (ray.PointAt(Stamp) - Center) / Radius;
		if (Stamp <= MaxDistance && Stamp >= MinDistance) {
			Hit.bHit = true;
			Hit.Stamp = Stamp;
			Hit.Normal = Normal;
			return Hit;
		}

		Stamp = (-b + Discriminant) / a;
		if (Stamp <= MaxDistance && Stamp >= MinDistance) {
			Hit.bHit = true;
			Hit.Stamp = Stamp;
			Hit.Normal = Normal;
			return Hit;
		}
	}
	
	return Hit;
}

template <unsigned char Bounces>
__device__ Vector4 CastRay(const Ray& ray, Vector4 * Spheres);

template <>
__device__ Vector4 CastRay<0>(const Ray& ray, Vector4 * Spheres);

template <unsigned char Bounces>
__device__ Vector4 CastRay(const Ray& ray, Vector4 * Spheres) {
	RayHit Hit1 = HitSphere(Spheres[0], Spheres[0].w, 0.001F, FLT_MAX, ray);
	RayHit Hit2 = HitSphere(Spheres[1], Spheres[1].w, 0.001F, FLT_MAX, ray);
	RayHit * Hit = (Hit1.bHit && Hit2.bHit) ? (Hit1.Stamp < Hit2.Stamp ? &Hit1 : &Hit2) : ( Hit1.bHit ? &Hit1 : &Hit2 );

	Vector3 Color = CastRay<0>(ray, Spheres);
	if (Hit->bHit) {
		// --- Normal Color ((Hit.Normal + 1.F) * 0.5F)
		Color = (Color + ( CastRay<Bounces - 1>(Ray(ray.PointAt(Hit->Stamp), Vector3::Reflect(ray.Direction(), Hit->Normal)), Spheres) ) ) * 0.5F;
	}

	return Color;
}

template <>
__device__ Vector4 CastRay<0>(const Ray& ray, Vector4 * Spheres) {
	Vector3 NormalizedDirection = ray.Direction().Normalized();
	float NHit = 0.5F * (NormalizedDirection.y + 1.F);
	Vector3 Color = Vector3(1.F) * (1.F - NHit) + Vector3(0.5F, 0.7F, 1.F) * NHit * 0.5F;
	return Color;
}

__global__ void InitRandomKernel(int2 TextureDimension, hiprandState * RandState) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= TextureDimension.x || y >= TextureDimension.y) return;

	int Index = y * TextureDimension.x + x;
	// --- Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, Index, 0, &RandState[Index]);
}

__global__ void WirteTextureKernel(int2 TextSize, Vector3 LowLeft, Vector3 Horizontal, Vector3 Vertical, Vector3 Origin, Vector4 * Spheres) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= TextSize.x || y >= TextSize.y) return;

	// int Index = y * TextureDimension.x + x;
	// float4 element = make_float4(0, 0, 0, 0);
	// surf2Dread(&element, SurfaceWrite, x * sizeof(float4), y);
	Vector4 Color = Vector4();
	Vector2 Coord; Ray ray;
	// for (int s = 0; s < 4; s++) {
		Coord.u = float(x + 0.25F) / float(TextSize.x);
		Coord.v = float(y + 0.1F) / float(TextSize.y);
		ray = Ray(Origin, LowLeft + (Horizontal * Coord.u) + (Vertical * Coord.v));
		Color += CastRay<2>(ray, Spheres);
		// Coord.u = float(x - 0.5) / float(TextSize.x);
		// Coord.v = float(y - 0.2) / float(TextSize.y);
		// ray = Ray(Origin, LowLeft + (Horizontal * Coord.u) + (Vertical * Coord.v));
		// Color += CastRay<2>(ray, Spheres);
	// }
	surf2Dwrite(Color / 1.0F, SurfaceWrite, x * sizeof(float4), y);
}

extern "C"
void LaunchWriteTextureKernel(hipArray *cudaTextureArray, int2 TextureDim, hiprandState * RandState, Vector4 * Spheres) {
	dim3 dimBlock(8, 8);
	dim3 dimGrid(TextureDim.x / dimBlock.x + 1, TextureDim.y / dimBlock.y + 1);

	// --- Bind texture array to a writable CUDA surface
	CUDA::Check( cudaBindSurfaceToArray(SurfaceWrite, cudaTextureArray) );

	// InitRandomKernel <<< dimGrid, dimBlock >>> (TextureDim, RandState);
	// CUDA::GetLastCudaError("InitRandomKernel Failed");
	// // --- Wait for GPU to finish
	// CUDA::Check( hipDeviceSynchronize() );

	float WidthRatio = TextureDim.x / 100.F;
	float HeightRatio = TextureDim.y / 100.F;
	WirteTextureKernel <<< dimGrid, dimBlock >>> (
		TextureDim, {-WidthRatio, -HeightRatio, -HeightRatio }, { 2 * WidthRatio, 0, 0}, {0, 2 * HeightRatio, 0}, 0, Spheres
	);
	CUDA::GetLastCudaError("WriteTextureKernel Failed");
	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );
}


int RayTracingTexture2D(Texture2D * texture, std::vector<Vector4> * Spheres) {
	hipGraphicsResource *cudaTextureResource;
	hipArray            *cudaTextureArray;
	hiprandState          *dRandState;
	Vector4              *dSpheres;

	IntVector2 TextureDim = texture->GetDimension();

	CUDA::Check( hipProfilerStart() );

	// --- Allocate Spheres
	CUDA::Check( hipMalloc((void **)&dSpheres, Spheres->size() * sizeof(Vector4)) );
	CUDA::Check( hipMemcpy(dSpheres, &(*Spheres)[0], Spheres->size() * sizeof(Vector4), hipMemcpyHostToDevice) );

	// --- Allocate pseudo random values
	CUDA::Check( hipMalloc((void **)&dRandState, TextureDim.x * TextureDim.y * sizeof(hiprandState)) );

	// --- Register Image (texture) to CUDA Resource
	CUDA::Check( hipGraphicsGLRegisterImage(&cudaTextureResource,
		texture->GetTextureObject(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore) 
	);

	Debug::Timer Timer;
	Timer.Start();
	// --- Map CUDA resource
	CUDA::Check( hipGraphicsMapResources(1, &cudaTextureResource, 0) );
	{
		// --- Get mapped array
		CUDA::Check( hipGraphicsSubResourceGetMappedArray(&cudaTextureArray, cudaTextureResource, 0, 0) );
		IntVector2 TextureDim = texture->GetDimension();
		LaunchWriteTextureKernel(cudaTextureArray, { TextureDim.x, TextureDim.y }, dRandState, dSpheres);
	}
	CUDA::Check( hipGraphicsUnmapResources(1, &cudaTextureResource, 0) );

	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );

	CUDA::Check( hipFree(dRandState) );

	Timer.Stop();
	Debug::Log(
		Debug::LogDebug, L"CUDA Texture Write/Read with total volume (%s): %dms",
		Text::FormatUnit(TextureDim.x * TextureDim.y, 3).c_str(),
		Timer.GetEnlapsed()
	);

	CUDA::Check( hipGraphicsUnregisterResource(cudaTextureResource) );

	return 0;
}