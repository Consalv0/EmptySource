#include "hip/hip_runtime.h"

#include "../include/Core.h"
#include "../include/CoreCUDA.h"
#include "../include/CoreTypes.h"

#include "../include/Graphics.h"
#include "../include/Texture2D.h"
#include "../include/Utility/Timer.h"
#include "../include/Math/CoreMath.h"
#include "../include/Mesh.h"

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <surface_functions.h>
#include <hip/hip_fp16.h>

surface<void, cudaSurfaceType2D> SurfaceWrite;

__device__ RayHit HitSphere(const Vector3& Center, const float& Radius, const float& MinDistance, const float& MaxDistance, const Ray& ray) {
	RayHit Hit;
	Vector3 OC = ray.Origin() - Center;
	float a = ray.Direction().Dot(ray.Direction());
	float b = OC.Dot(ray.Direction());
	float c = OC.Dot(OC) - Radius * Radius;
	float Discriminant = sqrtf((b * b) - (a * c));
	
	if (Discriminant >= 0.F) {
		float Stamp = (-b - Discriminant) / a;
		Vector3 Normal = (ray.PointAt(Stamp) - Center) / Radius;
		if (Stamp < MaxDistance && Stamp > MinDistance) {
			Hit.bHit = true;
			Hit.Stamp = Stamp;
			Hit.Normal = Normal;
			return Hit;
		}

		Stamp = (-b + Discriminant) / a;
		if (Stamp < MaxDistance && Stamp > MinDistance) {
			Hit.bHit = true;
			Hit.Stamp = Stamp;
			Hit.Normal = Normal;
			return Hit;
		}
	}
	
	return Hit;
}

__device__ Vector3 RandomSphericalDir(hiprandState LocalRandState) {
	Vector3 Direction;
	do {
		Direction = (Vector3(hiprand_uniform(&LocalRandState), hiprand_uniform(&LocalRandState), hiprand_uniform(&LocalRandState)) * 2.F) - Vector3(1, 1, 1);
	} while (Direction.MagnitudeSquared() >= 1.0);
	return Direction;
}

template <unsigned char Bounces>
__device__ Vector4 CastRay(const Ray& RayCast, Vector4 * Spheres, hiprandState LocalRandState);

template <>
__device__ Vector4 CastRay<0>(const Ray& RayCast, Vector4 * Spheres, hiprandState LocalRandState);

template <unsigned char Bounces>
__device__ Vector4 CastScreenRay(const Ray& RayCast, Vector4 * Spheres, hiprandState LocalRandState) {
	RayHit Hit1 = HitSphere(Spheres[0], Spheres[0].w, 0.001F, FLT_MAX, RayCast);
	RayHit Hit2 = HitSphere(Spheres[1], Spheres[1].w, 0.001F, Hit1.Stamp, RayCast);
	RayHit * Hit = (Hit1.bHit && Hit2.bHit) ? (Hit1.Stamp < Hit2.Stamp ? &Hit1 : &Hit2) : (Hit1.bHit ? &Hit1 : &Hit2);

	Vector3 Color = Vector3(0.F);
	if (Hit->bHit) {
		float Diffuse = 1;
		for (int i = 0; i < 1; i++) {
			Vector3 Target = RayCast.PointAt(Hit->Stamp) + Hit->Normal + RandomSphericalDir(LocalRandState) * 0.1F;
			// Target.Normalize();
			Diffuse *= 0.5F;
			// Color = ((Target + 1.F) * 0.5F);
			Color += CastRay<Bounces - 1>(Ray(RayCast.PointAt(Hit->Stamp), Target - RayCast.PointAt(Hit->Stamp)), Spheres, LocalRandState);
		}
		return Vector4(Color * Diffuse, 1);
	}

	return CastRay<0>(RayCast, Spheres, LocalRandState);
}

template <unsigned char Bounces>
__device__ Vector4 CastRay(const Ray& RayCast, Vector4 * Spheres, hiprandState LocalRandState) {
	RayHit Hit1 = HitSphere(Spheres[0], Spheres[0].w, 0.001F, FLT_MAX, RayCast);
	RayHit Hit2 = HitSphere(Spheres[1], Spheres[1].w, 0.001F, Hit1.Stamp, RayCast);
	RayHit * Hit = (Hit1.bHit && Hit2.bHit) ? (Hit1.Stamp < Hit2.Stamp ? &Hit1 : &Hit2) : ( Hit1.bHit ? &Hit1 : &Hit2 );

	Vector3 Color = Vector3(0.F);
	if (Hit->bHit) {
		Vector3 Target = RayCast.PointAt(Hit->Stamp) + Hit->Normal + RandomSphericalDir(LocalRandState) * 0.1F;
		// Target.Normalize();
		// Color = ((Target + 1.F) * 0.5F);
		Color += CastRay<Bounces - 1>(Ray(RayCast.PointAt(Hit->Stamp), Target - RayCast.PointAt(Hit->Stamp)), Spheres, LocalRandState);
		return Vector4(Color, 1);
	}

	return CastRay<0>(RayCast, Spheres, LocalRandState);
}

template <>
__device__ Vector4 CastRay<0>(const Ray& RayCast, Vector4 * Spheres, hiprandState LocalRandState) {
	Vector3 NormalizedDirection = RayCast.Direction().Normalized();
	float NHit = 0.5F * (NormalizedDirection.y + 1.F);
	Vector3 Color = Vector3(1.F) * (1.F - NHit) + Vector3(0.5F, 0.7F, 1.F) * NHit * 0.5F;
	return Vector4(Color, 1);
}

__global__ void WirteTextureKernel(
	int2 TextSize, 
	Vector3 LowLeft, 
	Vector3 Horizontal, Vector3 Vertical, 
	Vector3 Origin, 
	hiprandState * RandState, 
	Vector4 * Spheres) 
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= TextSize.x || y >= TextSize.y) return;
	int PixelIndex = y * TextSize.x + x;
	hiprandState LocalRandState = RandState[PixelIndex];

	// int Index = y * TextureDimension.x + x;
	// float4 element = make_float4(0, 0, 0, 0);
	// surf2Dread(&element, SurfaceWrite, x * sizeof(float4), y);
	Vector4 Color = Vector4();
	Vector2 Coord; Ray RayCast;
	const int Samples = 2;
	for (int s = 0; s < Samples; s++) {
		Coord.u = float(x + (hiprand_uniform(&LocalRandState) - 1) / 2.F) / float(TextSize.x);
		Coord.v = float(y + (hiprand_uniform(&LocalRandState) - 1) / 2.F) / float(TextSize.y);
		RayCast = Ray(Origin, LowLeft + (Horizontal * Coord.u) + (Vertical * Coord.v));
		Color += CastScreenRay<2>(RayCast, Spheres, LocalRandState);
	}
	Vector4 OutColor = { sqrtf(Color.x / Samples), sqrtf(Color.y / Samples), sqrtf(Color.z / Samples), Color.w / Samples};
	surf2Dwrite(OutColor, SurfaceWrite, x * sizeof(Vector4), y);
}

extern "C"
void LaunchWriteTextureKernel(hipArray *cudaTextureArray, int2 TextureDim, hiprandState * RandState, Vector4 * Spheres) {
	dim3 dimBlock(8, 8);
	dim3 dimGrid(TextureDim.x / dimBlock.x + 1, TextureDim.y / dimBlock.y + 1);

	// --- Bind texture array to a writable CUDA surface
	CUDA::Check( cudaBindSurfaceToArray(SurfaceWrite, cudaTextureArray) );

	float WidthRatio = TextureDim.x / 100.F;
	float HeightRatio = TextureDim.y / 100.F;
	WirteTextureKernel <<< dimGrid, dimBlock >>> (
		TextureDim, {-WidthRatio, -HeightRatio, -HeightRatio }, { 2 * WidthRatio, 0, 0}, {0, 2 * HeightRatio, 0}, 0, RandState, Spheres
	);
	CUDA::GetLastCudaError("WriteTextureKernel Failed");
	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );
}

int RTRenderToTexture2D(Texture2D * texture, std::vector<Vector4> * Spheres, const void * dRandState) {
	hipGraphicsResource *cudaTextureResource;
	hipArray            *cudaTextureArray;
	Vector4              *dSpheres;

	IntVector2 TextureDim = texture->GetDimension();

	CUDA::Check( hipProfilerStart() );

	// --- Allocate Spheres
	CUDA::Check( hipMalloc((void **)&dSpheres, Spheres->size() * sizeof(Vector4)) );
	CUDA::Check( hipMemcpy(dSpheres, &(*Spheres)[0], Spheres->size() * sizeof(Vector4), hipMemcpyHostToDevice) );

	// --- Register Image (texture) to CUDA Resource
	CUDA::Check( hipGraphicsGLRegisterImage(&cudaTextureResource,
		texture->GetTextureObject(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore) 
	);

	Debug::Timer Timer;
	Timer.Start();
	// --- Map CUDA resource
	CUDA::Check( hipGraphicsMapResources(1, &cudaTextureResource, 0) );
	{
		// --- Get mapped array
		CUDA::Check( hipGraphicsSubResourceGetMappedArray(&cudaTextureArray, cudaTextureResource, 0, 0) );
		IntVector2 TextureDim = texture->GetDimension();
		LaunchWriteTextureKernel(cudaTextureArray, { TextureDim.x, TextureDim.y }, (hiprandState*)dRandState, dSpheres);
	}
	CUDA::Check( hipGraphicsUnmapResources(1, &cudaTextureResource, 0) );

	// --- Wait for GPU to finish
	CUDA::Check( hipDeviceSynchronize() );

	Timer.Stop();
	Debug::Log(
		Debug::LogDebug, L"CUDA Texture Write/Read with total volume (%ls): %.2fms",
		Text::FormatUnit(TextureDim.x * TextureDim.y, 3).c_str(),
		Timer.GetEnlapsedMili()
	);

	CUDA::Check( hipGraphicsUnregisterResource(cudaTextureResource) );

	return 0;
}